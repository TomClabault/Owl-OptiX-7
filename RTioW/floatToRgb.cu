#include "hip/hip_runtime.h"
#include "viewer.h"

#include <>
#include <cuda_device_runtime_api.h>

void __global__ float4_to_rgb_kernel(float4* in, uint32_t* out, vec2i input_size)
{
    unsigned int pixel_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int pixel_y = blockIdx.y * blockDim.y + threadIdx.y;

    //Thread out of bounds
    if (pixel_x >= input_size.x || pixel_y >= input_size.y)
        return;

    unsigned int pixel_index = pixel_y * input_size.x + pixel_x;

    float4 f4 = in[pixel_index];
    vec3f gamma_corrected = vec3f(clamp(sqrtf(f4.x), 1.0f),
                                  clamp(sqrtf(f4.y), 1.0f),
                                  clamp(sqrtf(f4.z), 1.0f));

    uint32_t value = 0;
    value |= (uint32_t)(gamma_corrected.x * 255.9f) <<  0;
    value |= (uint32_t)(gamma_corrected.y * 255.9f) <<  8;
    value |= (uint32_t)(gamma_corrected.z * 255.9f) << 16;
    value |= (uint32_t)255             << 24;

    out[pixel_index] = value;
}

void Viewer::cuda_float4_to_rgb()
{
    vec2i block_size = 32;
    vec2i nb_blocks = divRoundUp(fbSize, block_size);

    if (denoiser_on)
        float4_to_rgb_kernel<<<dim3(nb_blocks.x, nb_blocks.y), dim3(block_size.x, block_size.y)>>>((float4*)m_denoised_buffer.d_pointer(), fbPointer, fbSize);
    else//If the denoiser isn't on, we still want to be able to update the frame so we're converting the non-denoised buffer
        float4_to_rgb_kernel<<<dim3(nb_blocks.x, nb_blocks.y), dim3(block_size.x, block_size.y)>>>((float4*)m_float_frame_buffer.d_pointer(), fbPointer, fbSize);
}
